#include "hip/hip_runtime.h"

__global__ void ExysSim(
        int numSims,
        Point* inputs, 
        Point* inputScratch,
        int inputSize,
        Point* observerScratch,
        int observerScratchSize,
        uint64_t execId, 
        uint64_t* execIdScratch)
{
    int tid = blockIdx.x;
    if(tid > numSims)
    {
        // no sim for me
        return;
    }
    
    // Point to my memory
    inputScratch += inputSize*tid;
    observerScratch += observerScratchSize*tid;
    execIdScratch += tid

    // Copy in inputs
    memcpy(inputs, inputScratch, inputSize*sizeof(double));

    CaptureStateBlock();

    StabilizeBlock(inputScratch, observerScratch);

    ResetBlock();
    
    // Check if our sim job is done

    // Run simfunc to update inputs

}
