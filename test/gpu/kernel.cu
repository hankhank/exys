#include "hip/hip_runtime.h"

#include "exys.h"

// Functions defined in the jitted code
namespace Exys {
 
extern "C" __device__ void (ExysStabilize)(Point* inputs, Point* observers);
extern "C" __device__ void (ExysCaptureState)();
extern "C" __device__ void (ExysResetState)();

__device__ int GetTid()
{
    return threadIdx.x + blockIdx.x * blockDim.x; 
}

__device__ bool RunBlock(int blocksInUse)
{
    return (GetTid() < blocksInUse);
}

__device__ void GetPtrsThisBlock(
        Point** inputScratch,
        int inputSize,
        Point** observerScratch,
        int observerScratchSize)
{
    int tid = GetTid();

    // Point to my memory
    if(inputScratch)    *inputScratch += inputSize*tid;
    if(observerScratch) *observerScratch += observerScratchSize*tid;
}

__device__ volatile uint64_t valRunCount = 0;

extern "C" __global__ void ExysVal(
        volatile uint64_t* inExecId, 
        volatile uint64_t* outExecId, 
        int numBlocksRunning,
        Point* inputs, 
        int inputSize,
        Point* observerScratch,
        int observerScratchSize)
{
    int tid = GetTid();

    if(!RunBlock(numBlocksRunning)) return;

    uint64_t curExecId = *inExecId;

    GetPtrsThisBlock(
            nullptr,
            0,
            &observerScratch,
            observerScratchSize);
    
    // Run this kernel hot hot hot
    while (true)
    {
        while(*inExecId && curExecId != *inExecId);
        
        ExysStabilize(inputs, observerScratch);

        ++curExecId;

        atomicAdd((unsigned long long int *)&valRunCount, 1);

        if(!tid) 
        {
            // all threads should be done and we are the master so update count
            while((valRunCount % numBlocksRunning) != 0);
            *outExecId = *inExecId;
        }
    }
}

extern "C" __global__ void ExysSim(
        volatile uint64_t* inExecId, 
        volatile uint64_t* outExecId, 
        int numBlocksRunning,
        Point* inputs, 
        Point* inputScratch,
        int inputSize,
        Point* observerScratch,
        int observerScratchSize)
{
    int tid = GetTid();

    if(!RunBlock(numBlocksRunning)) return;

    uint64_t curExecId = *inExecId;

    GetPtrsThisBlock(
            &inputScratch,
            inputSize,
            &observerScratch,
            observerScratchSize);
    
    // Run this kernel hot hot hot
    while (true)
    {
        while(*inExecId && curExecId != *inExecId);
        
        // Copy in inputs
        memcpy(inputs, inputScratch, inputSize*sizeof(Exys::Point));

        ExysCaptureState();

        ExysStabilize(inputScratch, observerScratch);

        ExysResetState();

        // Check if our sim job is done

        // Run simfunc to update inputs

        ++curExecId;

        atomicAdd((unsigned long long int *)&valRunCount, 1);

        if(!tid) 
        {
            // all threads should be done and we are the master so update count
            while((valRunCount % numBlocksRunning) != 0);
            *outExecId = *inExecId;
        }
    }

}

}
