#include "hip/hip_runtime.h"

#include "exys.h"

// Functions defined in the jitted code
namespace Exys {
 
extern "C" __device__ void (ExysStabilize)(Point* inputs, Point* observers);
extern "C" __device__ void (ExysCaptureState)();
extern "C" __device__ void (ExysResetState)();

__device__ int GetTid()
{
    return threadIdx.x + blockIdx.x * blockDim.x; 
}

__device__ bool RunBlock(int blocksInUse)
{
    return (GetTid() < blocksInUse);
}

__device__ void GetPtrsThisBlock(
        Point** inputScratch,
        int inputSize,
        Point** observerScratch,
        int observerScratchSize)
{
    int tid = GetTid();

    // Point to my memory
    if(inputScratch)    *inputScratch += inputSize*tid;
    if(observerScratch) *observerScratch += observerScratchSize*tid;
}

extern "C" __global__ void ExysVal(
        int numBlocksRunning,
        Point* inputs, 
        int inputSize,
        Point* observerScratch,
        int observerScratchSize)
{
    int tid = GetTid();

    if(!RunBlock(numBlocksRunning)) return;

    GetPtrsThisBlock(
            nullptr,
            0,
            &observerScratch,
            observerScratchSize);
    
    ExysStabilize(inputs, observerScratch);
}

extern "C" __global__ void ExysSim(
        int numBlocksRunning,
        Point* inputs, 
        Point* inputScratch,
        int inputSize,
        Point* observerScratch,
        int observerScratchSize,
        int sims)
{
    int tid = GetTid();

    if(!RunBlock(numBlocksRunning)) return;

    GetPtrsThisBlock(
            &inputScratch,
            inputSize,
            &observerScratch,
            observerScratchSize);
    
    // Copy in inputs
    memcpy(inputs, inputScratch, inputSize*sizeof(Exys::Point));
    
    for (int i = 0; i < sims; ++i)
    {
        ExysCaptureState();

        ExysStabilize(inputScratch, observerScratch);

        ExysResetState();
    }
}

}
