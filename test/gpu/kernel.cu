#include "hip/hip_runtime.h"

#include "exys.h"

// Functions defined in the jitted code
namespace Exys {

__device__ void (ExysStabilize)(Point* inputs, Point* observers);
__device__ void (ExysCaptureState)();
__device__ void (ExysResetState)();

__global__ void ExysSim(
        int numSims,
        Point* inputs, 
        Point* inputScratch,
        int inputSize,
        Point* observerScratch,
        int observerScratchSize,
        uint64_t execId, 
        uint64_t* execIdScratch)
{
    int tid = blockIdx.x;
    if(tid > numSims)
    {
        // no sim for me
        return;
    }
    
    // Point to my memory
    inputScratch += inputSize*tid;
    observerScratch += observerScratchSize*tid;
    execIdScratch += tid;

    // Copy in inputs
    memcpy(inputs, inputScratch, inputSize*sizeof(double));

    ExysCaptureState();

    ExysStabilize(inputScratch, observerScratch);

    ExysResetState();
    
    // Check if our sim job is done

    // Run simfunc to update inputs

}

}
